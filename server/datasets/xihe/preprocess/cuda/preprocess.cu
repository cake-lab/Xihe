
#include <hip/hip_runtime.h>
// CUDA kernels for accelerating data generation

__global__ void makePointCloud (
    float3 *dest, float *depthTexture, float2 *intrinsics
) {
    float2 f = intrinsics[0];
    float2 c = intrinsics[1];
    float2 res = intrinsics[2];

    int u = (blockIdx.x * blockDim.x) + threadIdx.x;
    int v = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (u > (res.x - 1) || v > (res.y - 1)) return;

    int linearIdx = v * res.x + u;

    float depth = depthTexture[linearIdx];

    float x = (u - c.x) * depth / f.x;
    float y = (v - c.y) * depth / f.y;
    float z = -depth;

    float3 position;
    position.x = x;
    position.y = y;
    position.z = z;

    dest[linearIdx] = position;
}

__global__ void cameraAdjustment (
  float3 *pointCloud,
  float *camToWorld, float *rotation, float2 *intrinsics
) {
  float2 res = intrinsics[2];
  int u = (blockIdx.x * blockDim.x) + threadIdx.x;
  int v = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (u > (res.x - 1) || v > (res.y - 1)) return;

  int linearIdx = v * res.x + u;

  float3 point = pointCloud[linearIdx];

  float3 position;

  position.x = camToWorld[0] * point.x + camToWorld[1] * point.y +\
   camToWorld[2] * point.z;
  position.y = camToWorld[4] * point.x + camToWorld[5] * point.y +\
   camToWorld[6] * point.z;
  position.z = camToWorld[8] * point.x + camToWorld[9] * point.y +\
   camToWorld[10] * point.z;

  float3 rPos;

  rPos.x = rotation[0] * position.x +\
    rotation[1] * position.y + rotation[2] * position.z;
  rPos.y = rotation[3] * position.x +\
    rotation[4] * position.y + rotation[5] * position.z;
  rPos.z = rotation[6] * position.x +\
    rotation[7] * position.y + rotation[8] * position.z;

  pointCloud[linearIdx] = rPos;
}

// Floating point numbers are not associative
// (a + b) + c != a + (b + c)
// running this function multiple times will result in
// different outputs, but the difference is acceptable.
__global__ void makeSHCoefficients (
    double3 *ldrDest, double3 *hdrDest, float *basis,
    float3 *cubemapColorLDR, float3 *cubemapColorHDR
) {
    int faceId = blockIdx.x;
    int pointId = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (pointId >= 128 * 128) return;

    int cubemapIdx = faceId * 128 * 128 + pointId;

    float3 ldrRgb = cubemapColorLDR[cubemapIdx];
    float3 hdrRgb = cubemapColorHDR[cubemapIdx];

    for (int i = 0; i < 9; i++)
    {
        int id = cubemapIdx * 9 + i;
        atomicAdd(&ldrDest[i].x, ldrRgb.x * basis[id]);
        atomicAdd(&ldrDest[i].y, ldrRgb.y * basis[id]);
        atomicAdd(&ldrDest[i].z, ldrRgb.z * basis[id]);

        atomicAdd(&hdrDest[i].x, hdrRgb.x * basis[id]);
        atomicAdd(&hdrDest[i].y, hdrRgb.y * basis[id]);
        atomicAdd(&hdrDest[i].z, hdrRgb.z * basis[id]);
    }
}
