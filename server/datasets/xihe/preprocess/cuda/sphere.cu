
#include <hip/hip_runtime.h>
#define CACHE_SIZE 1024
#define ANCHOR_SIZE 1280

__global__ void nn_search(
    float2 *dest, float3 *points, float3 *anchors
)
{
    const uint pointIdx = blockIdx.x;
    const uint anchorIdx = threadIdx.x;
    const uint threadSize = blockDim.x;

    float3 point = points[pointIdx];

    __shared__ uint idx[CACHE_SIZE];
    __shared__ float cosines[CACHE_SIZE];


    uint i = 0;
    uint rIdx = i * threadSize + anchorIdx;
    float pointLength = sqrt(point.x * point.x + point.y * point.y + point.z * point.z);

    // Initialize cache array
    // By default, CUDA shared array is uninitialized
    cosines[anchorIdx] = 0;

    // Compute angels for each anchor point and sphere point
    while (rIdx < ANCHOR_SIZE) {
        float3 anchor = anchors[rIdx];

        float p_cos = (point.x * anchor.x +\
            point.y * anchor.y + point.z * anchor.z) / pointLength;

        if (p_cos > cosines[anchorIdx]) {
            idx[anchorIdx] = rIdx;
            cosines[anchorIdx] = p_cos;
        }

        rIdx = ++i * threadSize + anchorIdx;
    }

    __syncthreads();

    // Begin reduction to find the maximum cosine (smallest angel)
    i = CACHE_SIZE / 2;

    while (i != 0) {
        if (anchorIdx < i) {
            rIdx = anchorIdx + i;
            if (cosines[anchorIdx] < cosines[rIdx]) {
                idx[anchorIdx] = idx[rIdx];
                cosines[anchorIdx] = cosines[rIdx];
            }
        }

        __syncthreads();

        i /= 2;
    }

    int selectedAnchorIdx = idx[0];

    atomicMin((int *)(&dest[selectedAnchorIdx].y), __float_as_int(pointLength));

    __syncwarp();

    if (dest[selectedAnchorIdx].y == pointLength) {
        dest[selectedAnchorIdx].x = pointIdx;
    }
}
